#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <iostream>
#include <getopt.h>
#include <cstdio>
#define checkError(err)                                                                \
  if ((err) != hipSuccess) {                                                          \
    printf("ERROR: %s in %s, line %d\n",hipGetErrorString(err), __FILE__, __LINE__);  \
    exit(EXIT_FAILURE);                                                                \
  }

using namespace cv;
using namespace std;

__device__
bool inside_image(int row, int col, int width, int height) {
  return row >= 0 && row < height && col >= 0 && col < width;
}

__global__
void convolutionKernel(unsigned char* image, float* kernel, float* out_image, int kernel_n, int width, int height) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (row < height && col < width) {
    int n = kernel_n / 2;
    float accumulation = 0;
    for (int i = -n; i <= n; i++) {
      for (int j = -n; j <= n; j++) {
        if (inside_image(row + i, col + j, width, height)) {
          int image_idx = (row + i) * width + (col + j);
          int kernel_idx = (n + i) * kernel_n + (n + j);
          accumulation += image[image_idx] * kernel[kernel_idx];
        }
      }
    }
    out_image[row * width + col] = accumulation;
  }
}

__global__
void magnitudeKernel(float* x, float* y, unsigned char* r, int width, int height) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < height && col < width) {
    int idx = row * width + col;
    r[idx] = (unsigned char) hypot(x[idx], y[idx]);
  }
}

void sobel(unsigned char *h_img, unsigned char *h_img_sobel, int width, int height) {
  unsigned char *d_img, *d_img_sobel;
  float *d_img_sobel_x, *d_img_sobel_y;
  float *d_sobel_x, *d_sobel_y;
  int size = width * height;
  hipError_t err;

  err = hipMalloc((void**) &d_img,         size * sizeof(unsigned char)); checkError(err);
  err = hipMalloc((void**) &d_img_sobel,   size * sizeof(unsigned char)); checkError(err);
  err = hipMalloc((void**) &d_img_sobel_x, size * sizeof(float));         checkError(err);
  err = hipMalloc((void**) &d_img_sobel_y, size * sizeof(float));         checkError(err);
  err = hipMalloc((void**) &d_sobel_x, 9 * sizeof(float)); checkError(err);
  err = hipMalloc((void**) &d_sobel_y, 9 * sizeof(float)); checkError(err);
  
  err = hipMemcpy(d_img, h_img, size * sizeof(unsigned char), hipMemcpyHostToDevice); checkError(err);

  float h_sobel_x[] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
  float h_sobel_y[] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
  err = hipMemcpy(d_sobel_x, h_sobel_x, 9 * sizeof(float), hipMemcpyHostToDevice); checkError(err);
  err = hipMemcpy(d_sobel_y, h_sobel_y, 9 * sizeof(float), hipMemcpyHostToDevice); checkError(err);

  int block_size = 32;
  dim3 dim_grid(ceil((double) width / block_size), ceil((double) height / block_size), 1);
  dim3 dim_block(block_size, block_size, 1);

  convolutionKernel<<<dim_grid, dim_block>>>(d_img, d_sobel_x, d_img_sobel_x, 3, width, height);
  hipDeviceSynchronize();
  
  convolutionKernel<<<dim_grid, dim_block>>>(d_img, d_sobel_y, d_img_sobel_y, 3, width, height);
  hipDeviceSynchronize();
  
  magnitudeKernel<<<dim_grid, dim_block>>>(d_img_sobel_x, d_img_sobel_y, d_img_sobel, width, height);
  hipDeviceSynchronize();

  err = hipMemcpy(h_img_sobel, d_img_sobel, size * sizeof(unsigned char), hipMemcpyDeviceToHost); checkError(err);
  
  err = hipFree(d_img); checkError(err);
  err = hipFree(d_img_sobel_x); checkError(err);
  err = hipFree(d_img_sobel_y); checkError(err);
  err = hipFree(d_img_sobel); checkError(err);
}

void runProgram(Mat& image, bool show) {
  int height = image.rows;
  int width = image.cols;
 
  unsigned char *img_sobel = (unsigned char*) malloc(width * height * sizeof(unsigned char));
  unsigned char *img = (unsigned char*) image.data;

  sobel(img, img_sobel, width, height);

  if (show) {
    imshow("Input", Mat(height, width, CV_8UC1, img));
    waitKey(0);
    imshow("Sobel operator", Mat(height, width, CV_8UC1, img_sobel));
    waitKey(0);
  }

  free(img_sobel);
}

void usage(char* program_name) {
  int n = 1;
  string opts[] = {"-s, --show"};
  string description[] = {
    "Show original image and result"
  };

  cout << "Usage: " << program_name << " [options ...] img1" << endl;
  cout << endl;
  cout << "Options" << endl;
  for (int i = 0; i < n; i++) {
    cout << "  " << opts[i] << ": " << description[i] << endl;
  }

  exit(EXIT_FAILURE);
}

int main(int argc, char** argv) {
  int opt, opt_index = 0;
    static struct option options[] = {
      {"show", no_argument, 0, 's'},
      {0, 0, 0, 0}
  };
  
  bool show = false;
  while ((opt = getopt_long(argc, argv, "s", options, &opt_index)) != -1) {
    switch (opt) {
      case 's':
        show = true;
        break;
      default:
        usage(argv[0]);
        break;
    }
  }
  
  if (argc - optind != 1) {
    cout << "Error: You must provide an image" << endl << endl;
    usage(argv[0]);
  }

  Mat image = imread(argv[optind], CV_LOAD_IMAGE_GRAYSCALE);
  if (!image.data) {
    printf("Could not open or find %s\n", argv[optind]);
    exit(EXIT_FAILURE);
  }
  
  runProgram(image, show);
  
  return 0;
}

