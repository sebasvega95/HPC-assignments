#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <iostream>
#include <getopt.h>
#include <cstdio>

#define BLUE 0
#define GREEN 1
#define RED 2

#define CHANNELS 3
#define GAMMA 2.2

using namespace cv;
using namespace std;

void checkError(hipError_t &err) {
  if (err != hipSuccess) {
    printf("ERROR: %s in %s, line %d\n",hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
}

__global__
void grayscaleKernel(unsigned char* d_img, unsigned char* d_out_img, int width, int height) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int idx = row * width + col;
  
  if (row < height && col < width) {
    float b = d_img[idx * CHANNELS + BLUE];
    float g = d_img[idx * CHANNELS + GREEN];
    float r = d_img[idx * CHANNELS + RED];
    float y = 0.2126 * powf(r / 255, GAMMA) + 0.7152 * powf(g / 255, GAMMA) + 0.0722 * powf(b / 255, GAMMA);
    d_out_img[idx] = MAX(0, 116 * powf(y, 1.0/3.0) - 16);
  }
}

void grayscale(unsigned char *h_img, unsigned char *img_grayscale, int width, int height) {
  int size = width * height * sizeof(unsigned char);
  unsigned char *d_img, *d_out_img;
  hipError_t err;
  
  err = hipMalloc((void**) &d_img, 3 * size); checkError(err);
  err = hipMalloc((void**) &d_out_img, size); checkError(err);
  err = hipMemcpy(d_img, h_img, 3 * size, hipMemcpyHostToDevice); checkError(err);
  
  int block_size = 32;
  dim3 dim_grid(ceil((double) width / block_size), ceil((double) height / block_size), 1);
  dim3 dim_block(block_size, block_size, 1);
  grayscaleKernel<<<dim_grid, dim_block>>>(d_img, d_out_img, width, height);
  hipDeviceSynchronize();
  err = hipMemcpy(img_grayscale, d_out_img, size, hipMemcpyDeviceToHost); checkError(err);
  
  err = hipFree(d_img); checkError(err);
  err = hipFree(d_out_img); checkError(err);
}

void runProgram(Mat& image, bool show) {
  int height = image.rows;
  int width = image.cols;
  int size = width * height * sizeof(unsigned char);
  
  unsigned char *img_grayscale = (unsigned char*) malloc(size);
  unsigned char *img = (unsigned char*) image.data;

  grayscale(img, img_grayscale, width, height);

  if (show) {
    imshow("Color", Mat(height, width, CV_8UC3, img));
    waitKey(0);
    imshow("Grayscale", Mat(height, width, CV_8UC1, img_grayscale));
    waitKey(0);
  }

  free(img_grayscale);
}

void usage(char* program_name) {
  int n = 1;
  string opts[] = {"-s, --show"};
  string description[] = {
    "Show original image and result"
  };

  cout << "Usage: " << program_name << " [options ...] img1" << endl;
  cout << endl;
  cout << "Options" << endl;
  for (int i = 0; i < n; i++) {
    cout << "  " << opts[i] << ": " << description[i] << endl;
  }

  exit(EXIT_FAILURE);
}

int main(int argc, char** argv) {
  int opt, opt_index = 0;
    static struct option options[] = {
      {"show", no_argument, 0, 's'},
      {0, 0, 0, 0}
  };
  
  bool show = false;
  while ((opt = getopt_long(argc, argv, "s", options, &opt_index)) != -1) {
    switch (opt) {
      case 's':
        show = true;
        break;
      default:
        usage(argv[0]);
        break;
    }
  }
  
  if (argc - optind != 1) {
    cout << "Error: You must provide an image" << endl << endl;
    usage(argv[0]);
  }

  Mat image = imread(argv[optind]);
  if(!image.data) {
    printf("Could not open or find %s\n", argv[optind]);
    exit(EXIT_FAILURE);
  }
  
  runProgram(image, show);
  
  return 0;
}

