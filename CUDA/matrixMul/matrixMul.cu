#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <getopt.h>
#include <random>
#include <cmath>
#include "matrix.hpp"

using namespace std;

void checkError(hipError_t &err) {
  if (err != hipSuccess) {
    printf("ERROR: %s in %s, line %d\n",hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
}

__global__
void matrixMulKernel(float *matA, float *matB, float *matC, int n) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < n && j < n) {
    float sum = 0;
    for (int k = 0; k < n; k++) {
      sum += matA[i * n + k] * matB[k * n + j];
    }
    matC[i * n + j] = sum;
  }
}

void matrixMul(float *h_matA, float *h_matB, float *h_matC, int n) {
  int size = n * n * sizeof(float);
  float *d_matA, *d_matB, *d_matC;
  hipError_t err;
  
  err = hipMalloc((void**) &d_matA, size); checkError(err);
  err = hipMalloc((void**) &d_matB, size); checkError(err);
  err = hipMalloc((void**) &d_matC, size); checkError(err);
  err = hipMemcpy(d_matA, h_matA, size, hipMemcpyHostToDevice); checkError(err);
  err = hipMemcpy(d_matB, h_matB, size, hipMemcpyHostToDevice); checkError(err);
  
  int block_size = 32;
  dim3 dim_grid(ceil((double) n / block_size), ceil((double) n / block_size), 1);
  dim3 dim_block(block_size, block_size, 1);
  matrixMulKernel<<<dim_grid, dim_block>>>(d_matA, d_matB, d_matC, n);
  hipDeviceSynchronize();
  err = hipMemcpy(h_matC, d_matC, size, hipMemcpyDeviceToHost); checkError(err);
  
  err = hipFree(d_matA); checkError(err);
  err = hipFree(d_matB); checkError(err);
  err = hipFree(d_matC); checkError(err);
}

float getMaxError(float *matA, float *matB, int n) {
  float max_error = -1;
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      float _error = fabs(matA[i * n + j] - matB[i * n + j]);
      max_error = max(max_error, _error);
    }
  }
  return max_error;
}

void runTest(float *matA, float *matB, float *d_matC, int n) {
  cout << "Finished parallel version, running sequential..." << endl;
  float *h_matC = (float*) malloc(n * n * sizeof(float));
  
  matrix::seqMul(matA, matB, h_matC, n);
  cout << "Done" << endl;
  
  float err = getMaxError(d_matC, h_matC, n);
  cout << "Max difference = " << err << endl;
}

void runProgram(int n, bool test) {
  float *matA = (float*) malloc(n * n * sizeof(float));
  float *matB = (float*) malloc(n * n * sizeof(float));
  float *matC = (float*) malloc(n * n * sizeof(float));
  
  matrix::initRandom(matA, n);
  matrix::initRandom(matB, n);
  
  matrixMul(matA, matB, matC, n);
  
  if (test) {
    runTest(matA, matB, matC, n);
  }
  
  free(matA);
  free(matB);
  free(matC);
}

void usage(char* program_name) {
  int n = 1;
  string opts[] = {"-t, --test"};
  string description[] = {
    "Test against sequential version"
  };

  cout << "Usage: " << program_name << " [options ...] num" << endl;
  cout << endl;
  cout << "Options" << endl;
  for (int i = 0; i < n; i++) {
    cout << "  " << opts[i] << ": " << description[i] << endl;
  }

  exit(EXIT_FAILURE);
}

int main(int argc, char** argv) {
  int opt, opt_index = 0;
    static struct option options[] = {
      {"test", no_argument, 0, 't'},
      {0, 0, 0, 0}
  };
  
  bool test = false;
  while ((opt = getopt_long(argc, argv, "t", options, &opt_index)) != -1) {
    switch (opt) {
      case 't':
        test = true;
        break;
      default:
        usage(argv[0]);
        break;
    }
  }
  
  if (argc - optind != 1) {
    cout << "Error: You must provide the size of the matrices" << endl << endl;
    usage(argv[0]);
  }

  int n = atoi(argv[optind]);
  runProgram(n, test);
  
  return 0;
}

